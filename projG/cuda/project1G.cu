#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <cmath>

__global__
void rgba_to_greyscale(uchar4* d_rgba, unsigned char* d_grey, int N)
{
    // Don't forget to check if the index is out of bounds
    // A simple `return` will break out for us

    // Suggest you use a static_cast when converting back to your
    // grey image's index

    // L = 0.21 ∗ Red + 0.72 ∗ Green + 0.07 ∗ Blue
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
       d_grey[idx] = 0.21 * d_rgba[idx].x + 0.72 * d_rgba[idx].y + 0.07 * d_rgba[idx].z; 
    }
    	
    return;
}

__global__ void
sobel_filter(unsigned char* d_grey, unsigned char* d_sobel, int rows, int cols)
{
    extern __shared__ uchar s_input2[];

    int sobelWidth = 3;
    int dx[3][3] = {-1, 0, 1,
                    -2, 0, 2,
                    -1, 0, 1};
    int dy[3][3] = { 1, 2, 1,
                     0, 0, 0,
                    -1,-2,-1};
    int sum_x = 0;
    int sum_y = 0;
    const int r = (sobelWidth - 1)/2;
    
    int block_size = blockDim.x*blockDim.y;
    int block_y = ((block_size)*gridDim.x)*blockIdx.y;
    int block_x = blockIdx.x*blockDim.x;
    int thread_y = (blockDim.x*gridDim.x)*threadIdx.y;
    int thread_x = threadIdx.x;
    int index = block_y + block_x + thread_y + thread_x;
    int shared_index = threadIdx.y *blockDim.x + threadIdx.x;

    // recheck this oob shit
    if (blockIdx.x*blockDim.x + threadIdx.x < rows && blockIdx.y*blockDim.y + threadIdx.y < cols) {
        s_input2[shared_index] = d_grey[index];
    } else {
        s_input2[shared_index] = 0;
    }

    __syncthreads();

    for (int r=-1; r<2; r++) {
        for (int c=-1; c<2; c++)
	{
	    int o_val = 0;
	    int sobel_y = r*(gridDim.x*blockDim.x);
	   
	   /* 
	    // if less/more than cached
	    if ((int)threadIdx.x + c < 0    || (int)threadIdx.y + r < 0 || (int)threadIdx.x + c > 15 || (int)threadIdx.y + r > 15) {
	        // if less than image
		if (block_y + thread_y + sobel_y < 0    || block_x + thread_x + c < 0 || block_y + thread_y + sobel_y > cols || block_x + thread_x + c > rows) {
		    o_val = 0;
		} else {
		// if still in bounds of image, grab val from image
		   int aug_index = index + r*(blockDim.x*gridDim.x) + c;
		    o_val =  d_grey[aug_index];
		}	
            } else {
	    // in bounds of cache, grab from cache
	    	o_val = s_input2[shared_index + sobel_y + c];
	    }
	*/
	    o_val = s_input2[shared_index + sobel_y + c];
	    sum_x = sum_x + o_val * dx[1+r][1+c];
	    sum_y = sum_y + o_val * dy[1+r][1+c];
	}
    }

    // Set your output to = (uchar)(abs(sum_x)+abs(sum_y));
    d_sobel[index] = (unsigned char)(abs(sum_x)+abs(sum_y));
}

void
your_rgba_to_greyscale(uchar4* d_rgba, unsigned char* d_grey, int rows, int cols) {
    // going to somewhat arbitrarily use 128 threads per block
    // only doing one calculation per pixel, keeping 1 dimensional
    // pixels represennted as 1d array
    // use formula (N + M-1) / M to calc grid size

    // **********************************
    // [    ][    ][    ][     ][    ][    ]

    // there will be indexing issues, account for in kernel

    int N = rows*cols;
    int M = 128;
    const dim3 blockSize(M);
    const dim3 gridSize( (N + M-1)/M );
    rgba_to_greyscale<<<gridSize,blockSize>>>(d_rgba, d_grey, N);
    hipDeviceSynchronize();
}

void
your_sobel(unsigned char* d_grey, unsigned char* d_sobel, int rows, int cols)
{
    int N = rows*cols;
    int M = 16;
    const dim3 blockDimHist(M,M,1);
    const dim3 gridDimHist( (rows + M-1)/M, (cols + M-1)/M );
    size_t blockSharedMemory = blockDimHist.x*blockDimHist.y*sizeof(uchar);
    sobel_filter<<<gridDimHist, blockDimHist, blockSharedMemory>>>(d_grey, d_sobel, rows, cols);
    hipDeviceSynchronize(); // maybe delete this
}

int main(int argc, char **argv)
{
    uchar4        *h_rgbaImage, *d_rgbaImage;
    unsigned char *h_greyImage, *d_greyImage;
    unsigned char *h_sobel, *d_sobel;
    std::string input_file;
    std::string greyscale_file;
    std::string sobel_file;

    switch(argc)
    {
        
	case 2:
            input_file = std::string(argv[1]);
            greyscale_file = "project1G_greyscale.png";
            sobel_file = "project1G_sobel.png";
            break;
        case 3:
            input_file = std::string(argv[1]);
            greyscale_file = std::string(argv[2]);
            sobel_file = "project1G_sobel.png";
            break;
        case 4:
            input_file = std::string(argv[1]);
            greyscale_file = std::string(argv[2]);
            sobel_file = std::string(argv[3]);
            break;
        default:
            std::cerr << "Usage: ./project1G input_file [greyscale_filename]" 
                      << "[sobel_filename]"
                      << std::endl;
            exit(1);
    }

    cv::Mat image;
    image = cv::imread(input_file.c_str(), CV_LOAD_IMAGE_COLOR);
    if(image.empty())
    {
        std::cerr << "Couldn't open file: " << input_file << std::endl;
        exit(1);
    }   
    cv::Mat imageRGBA;
    cv::Mat imageGrey;
    cv::Mat imageSobel;
    cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

    imageGrey.create(image.rows, image.cols, CV_8UC1);
    imageSobel.create(image.rows, image.cols, CV_8UC1);

    if(!imageRGBA.isContinuous() || !imageGrey.isContinuous())
    {
        std::cerr << "Images aren't continuous. Exiting" << std::endl;
        exit(1);
    }

    *(&h_rgbaImage) = (uchar4*)imageRGBA.ptr<unsigned char>(0);
    *(&h_greyImage) = imageGrey.ptr<unsigned char>(0);
    *(&h_sobel) = imageSobel.ptr<unsigned char>(0);
    size_t numRows = imageRGBA.rows;
    size_t numCols = imageRGBA.cols;
    size_t numPixels = numRows * numCols;

    // Allocate all your memory here. Use cudaMallocs, Memset, and Memcpy
    //std::cout << "Input Image: R x C: " << numRows << " x " << numCols << std::endl;

    // print rgba image pixel vals
    //std::cout << "RGBA Value Matrix:" << std::endl;
    /*
    for (int i = 1; i <= numPixels; i++) {

	    std::cout << "(" << static_cast<unsigned>(h_rgbaImage[i-1].x) << " "
		      << static_cast<unsigned>(h_rgbaImage[i-1].y) << " "
		      << static_cast<unsigned>(h_rgbaImage[i-1].z) << " "
		      << static_cast<unsigned>(h_rgbaImage[i-1].w) << ") ";
    	    if (i % numCols == 0) {
	    	std::cout << std::endl;
	    }
    }
    */

	
    // device hipMalloc for rgba and grey images
    hipMalloc( (uchar4 **)&d_rgbaImage, sizeof(uchar4)*numPixels );
    hipMalloc( (unsigned char **)&d_greyImage, sizeof(unsigned char)*numPixels );

    // host to device hipMemcpy for rgba image
    hipMemcpy(d_rgbaImage, h_rgbaImage, sizeof(uchar4)*numPixels, hipMemcpyHostToDevice);

    your_rgba_to_greyscale(d_rgbaImage, d_greyImage, numRows, numCols);
    hipDeviceSynchronize();

    // Do a memcpy for your grey Image here (Device to Host)
    hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char)*numPixels, hipMemcpyDeviceToHost);

    // print rgba image pixel vals
    /*
    std::cout << "GREY Value Matrix:" << std::endl;
    for (int i = 1; i <= numPixels; i++) {

	    std::cout << "(" << static_cast<unsigned>(h_greyImage[i-1]) << ") ";
    	    if (i % numCols == 0) {
	    	std::cout << std::endl;
	    }
    }
    */

    // Write out greyscale
    cv::Mat output(numRows, numCols, CV_8UC1, (void*)h_greyImage);
    cv::imwrite(greyscale_file.c_str(), output);

    hipDeviceSynchronize();

    // Now we'll do the Sobel filter

    // hipMalloc for sobel output
    hipMalloc( (unsigned char**)&d_sobel, sizeof(unsigned char)*numPixels);

    your_sobel(d_greyImage, d_sobel, numRows, numCols);

    // device to host memcpy of sobel
    hipMemcpy(h_sobel, d_sobel, sizeof(unsigned char)*numPixels, hipMemcpyDeviceToHost);

    // Write Sobel
    cv::Mat output2(numRows, numCols, CV_8UC1, (void*)h_sobel);
    cv::imwrite(sobel_file.c_str(), output2);

    // Free \o/
    hipFree(d_rgbaImage);
    hipFree(d_greyImage);
    hipFree(d_sobel);
    return 0;
}
